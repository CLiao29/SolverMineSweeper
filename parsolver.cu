#include "hip/hip_runtime.h"
#include <stdio.h>
#include <tuple>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"
#include "game.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


using namespace std;

float toBW(int bytes, float sec) {
    return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
  }

__device__ float generate(hiprandState* globalState, int ind)
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__device__ void chooseRandomMove(int* playboard, int* board, int height, int width, hiprandState* globalState, int* resx, int* resy) {
    printf("IN CHOOSE RM\n");
    *resx = int(generate(globalState, 0) * height);
    *resy = int(generate(globalState, 0) * width);
    printf("in crm: %d %d\n",*resx, *resy);
    while (playboard[*resx * width + *resy] == 1) {
        *resx = (int)(generate(globalState, 0) * height);
        *resy = (int)(generate(globalState, 0) * width);
    }
}

//count uncovered adj mines (meaning already marked)
__device__ void countAdjMines(int* playboard, int* board, int height, int width, int x, int y, int* res) {
    int c = 0;
    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            int xi = x+i;
            int yi = y+j;
            if (xi >= 0 && xi < height && yi >= 0 && yi < width && !(i == 0 && j == 0)) {
                if (playboard[xi * width + yi] == 1 && board[xi * width + yi] == -1) {
                    c++;
                } 
            }
        }
    } 
    *res = c;
}

//reveal neighbors not revealed yet
__device__  void revealNeighbors(int* playboard, int* board, int height, int width, int x, int y) {
    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            int xi = x+i;
            int yi = y+j;
            if (xi >= 0 && xi < height && yi >= 0 && yi < width && !(i == 0 && j == 0)) {
                if (playboard[xi * width + yi] == 0) {
                    playboard[xi * width + yi] = 1;
                    if (board[xi * width + yi] == -1) {
                        printf("DID A BAD\n");
                    }
                } 
            }
        }
    } 
}

__device__  void countUnrevealed(int* playboard, int* board, int height, int width, int x, int y, int* res) {
    int c = 0;
    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            int xi = x+i;
            int yi = y+j;
            if (xi >= 0 && xi < height && yi >= 0 && yi < width && !(i == 0 && j == 0)) {
                if (playboard[xi * width + yi] == 0) {
                    c++;
                } 
            }
        }
    } 
    *res = c;
}
__device__  void markNeighbors(int* playboard, int* board, int height, int width, int* device_result, int* minesFound, int x, int y) {
    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            int xi = x+i;
            int yi = y+j;
            if (xi >= 0 && xi < height && yi >= 0 && yi < width && !(i == 0 && j == 0)) {
                if (playboard[xi * width + yi] == 0) {
                    playboard[xi * width + yi] = 1;
                    //TODO: make atomic
                    device_result[*minesFound*2] = xi;
                    device_result[*minesFound*2 + 1] = yi;
                    (*minesFound)++;
                }
            }
        }
    } 
    
}

__global__ void parSolveKernel(int* device_board, int* device_playboard, int* device_result, int* minesFound, int height, int width, int numMines, hiprandState* globalState) {
    printf("IN PARSOLVEKERNEL\n");
    int guesses = 0;  
    while(*minesFound < numMines) {
        int x, y;
        printf("IN THE WHOLE LOO\n");
        chooseRandomMove(device_playboard, device_board, height, width, globalState, &x, &y);
        guesses++;
        printf("%d %d\n",x,y);
        if (device_board[x * width + y] == -1) {
            printf("\n");
            printf("oops %dth guess was a bomb big sad\n",guesses);
            return;
        } else {
            //reveal
            device_playboard[x * width + y] = 1;
        }
        bool progress = true;
        while (progress) {
            progress = false;
            for (int i = 0; i < height; i++) {
                for (int j = 0; j < width; j++) {
                    if (device_playboard[i * width + j] == 1 && device_board[i * width + j] != -1 ) { //clear square
                        int adjmines;
                        countAdjMines(device_playboard, device_board, height, width, i,j, &adjmines);
                        int unrevealed;
                        countUnrevealed(device_playboard, device_board, height, width, i,j, &unrevealed);
                        if (unrevealed != 0 ){
                            if (adjmines == device_board[i * width + j]) { //all mines found
                                //reveal neighbors
                                progress = true;
                                revealNeighbors(device_playboard, device_board, height, width, i,j);
                            }
                            if (unrevealed == device_board[i * width + j] - adjmines && unrevealed >= 0) {
                                progress = true;
                                markNeighbors(device_playboard, device_board, height, width, device_result, minesFound, i,j);
                            }
                        }
                        
                    }
                }
            }
        }
    }
}

__global__ void setup_kernel( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}


void Game::parSolve() {

    int totalBytes = sizeof(int) * height * width;

    // compute number of blocks and threads per block
    // const int threadsPerBlock = 512;
    // const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int threadsPerBlock = 1;
    const int blocks = 1;
    // int N = width * height;

    int* device_board;
    int* device_playboard;
    int* device_result;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //

    printf("01\n");
    int* minesfound;
    hipMalloc(&minesfound,sizeof(int));
    hipMalloc(&device_board,sizeof(int)*height*width);
    hipMalloc(&device_playboard,sizeof(int)*height*width);
    hipMalloc(&device_result,sizeof(int)*numMines*2);
    printf("02\n");




    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //
    

    hipMemcpy(device_board,parboard,sizeof(int)*width*height,hipMemcpyHostToDevice);
    hipMemcpy(device_playboard,parplayboard,sizeof(int)*width*height,hipMemcpyHostToDevice);
    hipMemcpy(device_result,parplaymines,sizeof(int)*numMines*2,hipMemcpyHostToDevice);

    printf("04\n");

    double startTimeKernel = CycleTimer::currentSeconds();
    // run kernel

    //random
    hiprandState* devStates;
    printf("1\n");
    hipMalloc (&devStates, width * height * sizeof(hiprandState));
    srand(time(0));
    int seed = rand();
    printf("2\n");
    setup_kernel<<<blocks, threadsPerBlock>>>(devStates,seed);
    printf("3\n");
    parSolveKernel<<<blocks, threadsPerBlock>>>(device_board, device_playboard, device_result, minesfound, height, width, numMines, devStates);
    printf("4\n");

    hipDeviceSynchronize();
    double endTimeKernel = CycleTimer::currentSeconds(); 

    //
    // TODO copy result from GPU using hipMemcpy
    //


    hipMemcpy(parplaymines,device_result,sizeof(int)*2*numMines,hipMemcpyDeviceToHost);



    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));
    double kernelDuration = endTimeKernel - startTimeKernel;
    printf("Kernel: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * kernelDuration, toBW(totalBytes, kernelDuration));

    // TODO free memory buffers on the GPU
    hipFree(device_board);
    hipFree(device_playboard);
    hipFree(device_result);

}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}